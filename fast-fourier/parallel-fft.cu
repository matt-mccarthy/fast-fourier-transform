#include "hip/hip_runtime.h"
// This program takes n, block count, thread count, and number of trials as parameters.
// It is assumed that these are all powers of 2.
// Compiles with nvcc -std=c++11 -rdc=true -arch=compute_50 -code=sm_50
#include <chrono>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <utility>

#include "include/fast-fourier.h"

using namespace std;
using namespace chrono;
using namespace fast_fourier;

void	gen_array(cfloat* output, int n);
long double	sum(long double* in, int n);
long double	std_dev(long double* in, int n, long double sum);

int main(int argc, char** argv)
{
	if (argc < 3)
	{
		cerr << "Usage is " << argv[0] << " n num_trials num_blk num_thd" << endl;
		return 1;
	}

	int		n(atoi(argv[1]));
	int		trial_count(atoi(argv[2]));
	int		num_blk(atoi(argv[3]));
	int		num_thd(atoi(argv[4]));

	cfloat*	input(new cfloat[n]);
	cfloat* output(new cfloat[n]);
	cfloat* d_input(nullptr);
	cfloat* d_output(nullptr);
	bool*	binary_stor(nullptr);

	long double	times[trial_count];
	high_resolution_clock::time_point tp2, tp1;
	duration<long double> time_span;

	// Allocate device arrays
	if (hipMalloc( &d_input, sizeof(cfloat) * n ) != hipSuccess)
	{
		auto t = hipGetLastError();
		cout << "Failed to allocate input: "
			<< hipGetErrorName(t) << ", "
			<< hipGetErrorString(t) << endl;
		return 1;
	}
	if (hipMalloc( &d_output, sizeof(cfloat) * n ) != hipSuccess)
	{
		auto t = hipGetLastError();
		cout << "Failed to allocate output: "
			<< hipGetErrorName(t) << ", "
			<< hipGetErrorString(t) << endl;
		return 1;
	}
	if (hipMalloc( &binary_stor, sizeof(bool) * ilogbf(n) * num_thd * num_blk) != hipSuccess)
	{
		auto t = hipGetLastError();
		cout << "Failed to allocate boolean storage: "
			<< hipGetErrorName(t) << ", "
			<< hipGetErrorString(t) << endl;
		return 1;
	}

	// Run experiment
	for (int j(0) ; j < trial_count ; j++)
	{
		// Generate random input
		gen_array(input, n);

		// Run the test
		tp1 = system_clock::now();
		// Copy the input array to the GPU
		if (hipMemcpy( d_input, input, (long) n * sizeof(cfloat), hipMemcpyHostToDevice ) != hipSuccess)
		{
			auto t = hipGetLastError();
			cout << "Iteration: " << j
				<< " Input failed to copy: "
				<< hipGetErrorName(t) << ", "
				<< hipGetErrorString(t) << endl;
			return 1;
		}
		fast_fourier_transform<<<1,1>>>(d_input, d_output, n, num_blk, num_thd, binary_stor);
		if (hipMemcpy( output, d_output, (long) n * sizeof(cfloat), hipMemcpyDeviceToHost ) != hipSuccess)
		{
			auto t = hipGetLastError();
			cout << "Iteration: " << j
				<< " Output failed to copy: "
				<< hipGetErrorName(t) << ", "
				<< hipGetErrorString(t) << endl;
			return 1;
		}
		tp2 = system_clock::now();

		time_span	= duration_cast< duration<long double> >(tp2 - tp1)*1000.0;
		times[j]	= time_span.count();
	}

	// Calculate statistics
	long double av(sum(times, trial_count));
	long double sd(std_dev(times, trial_count, av));
	av /= (long double)n;

	cout << av << "\t" << sd << endl;

	hipFree( binary_stor );
	hipFree( d_input );
	hipFree( d_output );
	return 0;
}

void gen_array(cfloat* output, int n)
{
	srand(time(nullptr));

	for (int j = 0; j < n; j++)
         output[j] = cfloat(rand(), rand());
}

long double	sum(long double* in, int n)
{
	long double s(0.0);

	for (int j(0) ; j < n ; j++)
		s += in[j];

	return s;
}

long double	std_dev(long double* in, int n, long double sum)
{
	long double var = 0;
	long double tmp = 0;

	for (int i = 0 ; i < n ; i++)
	{
		tmp = (n * in[i] - sum);
		var += tmp * tmp;
	}

	long double stdDev = sqrt(var/n) / n;

	return stdDev;
}
