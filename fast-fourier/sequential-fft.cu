#include "hip/hip_runtime.h"
// This program takes n and trial count as parameters and nothing more.
// It is assumed that n is a power of 2.
// Compiles with nvcc -std=c++11 -rdc=true -arch=compute_50 -code=sm_50
#include <chrono>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <utility>

#include "include/fast-fourier.h"

using namespace std;
using namespace chrono;
using namespace fast_fourier;

void	gen_array(cfloat* output, int n);
long double	sum(long double* in, int n);
long double	std_dev(long double* in, int n, long double sum);

__global__
void run_test(cfloat* input, cfloat* output, int n)
{
	fast_fourier_transform(input, output, n);
}

int main(int argc, char** argv)
{
	if (argc < 3)
	{
		cerr << "Usage is " << argv[0] << " n num_trials" << endl;
		return 1;
	}

	int		n(atoi(argv[1]));
	int		trial_count(atoi(argv[2]));

	cfloat	input[n];
	cfloat* d_input(nullptr);
	cfloat* d_output(nullptr);

	long double	times[trial_count];
	high_resolution_clock::time_point tp2, tp1;
	duration<long double> time_span;

	// Allocate two device arrays
	if (hipMalloc( &d_input, sizeof(cfloat) * n ) != hipSuccess)
	{
		auto t = hipGetLastError();
		cout << "Failed to allocate input: "
			<< hipGetErrorName(t) << ", "
			<< hipGetErrorString(t) << endl;
		return 1;
	}
	if (hipMalloc( &d_output, sizeof(cfloat) * n ) != hipSuccess)
	{
		auto t = hipGetLastError();
		cout << "Failed to allocate output: "
			<< hipGetErrorName(t) << ", "
			<< hipGetErrorString(t) << endl;
		return 1;
	}

	// Run experiment
	for (int j(0) ; j < trial_count ; j++)
	{
		// Generate random input
		gen_array(input, n);

		// Copy the input array to the GPU
		if (hipMemcpy( d_input, input, sizeof(cfloat) * n, hipMemcpyHostToDevice ) != hipSuccess)
		{
			auto t = hipGetLastError();
			cout << "Input failed to copy: "
				<< hipGetErrorName(t) << ", "
				<< hipGetErrorString(t) << endl;
			return 1;
		}

		// Run the test
		tp1 = system_clock::now();
		run_test<<<1,1>>>(d_input, d_output, n);
		tp2 = system_clock::now();

		time_span	= duration_cast< duration<long double> >(tp2 - tp1)*1000.0;
		times[j]	= time_span.count();
	}

	// Calculate statistics
	long double av(sum(times, trial_count));
	long double sd(std_dev(times, trial_count, av));
	av /= (long double)n;

	cout << av << "\t" << sd << endl;

	hipFree( d_input );
	hipFree( d_output );
	return 0;
}

void gen_array(cfloat* output, int n)
{
	srand(time(nullptr));

	for (int j = 0; j < n; j++)
         output[j] = cfloat(rand(), rand());
}

long double	sum(long double* in, int n)
{
	long double s(0.0);

	for (int j(0) ; j < n ; j++)
		s += in[j];

	return s;
}

long double	std_dev(long double* in, int n, long double sum)
{
	long double var = 0;
	long double tmp = 0;

	for (int i = 0 ; i < n ; i++)
	{
		tmp = (n * in[i] - sum);
		var += tmp * tmp;
	}

	long double stdDev = sqrt(var/n) / n;

	return stdDev;
}
