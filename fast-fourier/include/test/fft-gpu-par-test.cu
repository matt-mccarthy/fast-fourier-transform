#include <iostream>

#include <thrust/complex.h>

#include "../fast-fourier.h"

using namespace std;
using namespace fast_fourier;

int main()
{
	int		n(16);
	int		num_blk(2), num_thd(4);
	cfloat	input[]		= {1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16};
	cfloat*	expected	= discrete_fourier_transform(input, n);

	cfloat*	d_input(nullptr);
	cfloat*	d_actual(nullptr);
	cfloat	actual[n];

	// Allocate an input and output array on the GPU
	if (hipMalloc( &d_input, sizeof(cfloat) * n ) != hipSuccess)
	{
		auto t = hipGetLastError();
		cout << "Failed to allocate input: "
			<< hipGetErrorName(t) << ", "
			<< hipGetErrorString(t) << endl;
		return 1;
	}
	if (hipMalloc( &d_actual, sizeof(cfloat) * n ) != hipSuccess)
	{
		auto t = hipGetLastError();
		cout << "Failed to allocate output: "
			<< hipGetErrorName(t) << ", "
			<< hipGetErrorString(t) << endl;
		return 1;
	}
	// Copy the input array to the GPU
	if (hipMemcpy( d_input, input, sizeof(cfloat) * n, hipMemcpyHostToDevice ) != hipSuccess)
	{
		auto t = hipGetLastError();
		cout << "Input failed to copy: "
			<< hipGetErrorName(t) << ", "
			<< hipGetErrorString(t) << endl;
		return 1;
	}

	fast_fourier_transform<<<1,1>>>(d_input, d_actual, n, num_blk, num_thd);

	// Copy the output array from the GPU
	if (hipMemcpy( actual, d_actual, sizeof(cfloat) * n, hipMemcpyDeviceToHost ) != hipSuccess)
	{
		auto t = hipGetLastError();
		cout << "Output failed to copy: "
			<< hipGetErrorName(t) << ", "
			<< hipGetErrorString(t) << endl;
		return 1;
	}

	for (int j(0) ; j < n ; j++)
		cout << actual[j] << "\t\t\t\t" << expected[j] << endl;

	delete[] expected;
	hipFree( d_actual );
	hipFree( d_input );

	return 0;
}
