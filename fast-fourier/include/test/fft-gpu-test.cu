#include "hip/hip_runtime.h"
#include <iostream>

#include <thrust/complex.h>
#include "../fast-fourier.h"

using namespace std;
using namespace fast_fourier;

__global__
void run_test(cfloat* input, cfloat* output, int n)
{
	cfloat* tmp(fast_fourier_transform(input, n));

	for (int j(0) ; j < n ; j++)
		output[j] = tmp[j];

	delete[] tmp;
}

int main()
{
	cfloat	input[]		= {1,2,3,4,5,6,7,8};
	cfloat*	expected	= discrete_fourier_transform(input, 8);

	cfloat* d_input, *d_actual, *actual;

	if (hipMalloc( &d_input, sizeof(cfloat) * 8 ) != hipSuccess)
	{
		cout << "Failed to allocate input" << endl;
		return 1;
	}
	if (hipMalloc( &d_actual, sizeof(cfloat) * 8 ) != hipSuccess)
	{
		cout << "Failed to allocate output" << endl;
		return 1;
	}
	if (hipMemcpy( d_input, input, sizeof(cfloat) * 8, hipMemcpyHostToDevice ) != hipSuccess)
	{
		cout << "Input failed to copy" << endl;
		return 1;
	}
	run_test<<<1,1>>>(d_input, d_actual, 8);

	actual		= new cfloat[8];
	hipMemcpy( actual, d_actual, sizeof(cfloat) * 8, hipMemcpyDeviceToHost );

	for (int j(0) ; j < 8 ; j++)
		cout << actual[j] << "\t\t\t\t" << expected[j] << endl;

	cout << "a" << endl;
	delete[] actual;
	cout << "b" << endl;
	delete[] d_actual;
	cout << "c" << endl;
	hipFree( d_input );

	return 0;
}
