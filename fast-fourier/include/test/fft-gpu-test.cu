#include "hip/hip_runtime.h"
#include <iostream>

#include <thrust/complex.h>

#include "../fast-fourier.h"

using namespace std;
using namespace fast_fourier;

__global__
void run_test(cfloat* input, cfloat* output, int n)
{
	fast_fourier_transform(input, output, n);
}

int main()
{
	int		n(8);
	cfloat	input[]		= {1,2,3,4,5,6,7,8};
	cfloat*	expected	= discrete_fourier_transform(input, n);

	cfloat*	d_input(nullptr);
	cfloat*	d_actual(nullptr);
	cfloat	actual[n];

	// Allocate an input and output array on the GPU
	if (hipMalloc( &d_input, sizeof(cfloat) * n ) != hipSuccess)
	{
		auto t = hipGetLastError();
		cout << "Failed to allocate input: "
			<< hipGetErrorName(t) << ", "
			<< hipGetErrorString(t) << endl;
		return 1;
	}
	if (hipMalloc( &d_actual, sizeof(cfloat) * n ) != hipSuccess)
	{
		auto t = hipGetLastError();
		cout << "Failed to allocate output: "
			<< hipGetErrorName(t) << ", "
			<< hipGetErrorString(t) << endl;
		return 1;
	}
	// Copy the input array to the GPU
	if (hipMemcpy( d_input, input, sizeof(cfloat) * n, hipMemcpyHostToDevice ) != hipSuccess)
	{
		auto t = hipGetLastError();
		cout << "Input failed to copy: "
			<< hipGetErrorName(t) << ", "
			<< hipGetErrorString(t) << endl;
		return 1;
	}

	run_test<<<1,1>>>(d_input, d_actual, n);

	// Copy the output array from the GPU
	if (hipMemcpy( actual, d_actual, sizeof(cfloat) * n, hipMemcpyDeviceToHost ) != hipSuccess)
	{
		auto t = hipGetLastError();
		cout << "Output failed to copy: "
			<< hipGetErrorName(t) << ", "
			<< hipGetErrorString(t) << endl;
		return 1;
	}

	for (int j(0) ; j < n ; j++)
		cout << actual[j] << "\t\t\t\t" << expected[j] << endl;

	hipFree( d_actual );
	hipFree( d_input );

	return 0;
}
